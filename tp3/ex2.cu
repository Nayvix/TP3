
#include <hip/hip_runtime.h>
#include <iostream>

//
// example: CUDA_CHECK( cudaMalloc(dx, x, N*sizeof(int) );
//
#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        std::cout << file << ':' << line << ": [CUDA ERROR] " << hipGetErrorString(code) << std::endl; 
        std::abort();
    }
}

constexpr int bloc_count       = 128; // constexpr equivalent to blockDim.x in CUDA kernel
constexpr int threads_per_bloc = 32;  // constexpr equivalent to gridDim.x  in CUDA kernel

constexpr int B = bloc_count;
constexpr int T = threads_per_bloc;

//
// step 04
//
// dx: array of size N
// dy: array of size N
// dz: array of size B
//






int main()
{
    constexpr int N = 1e6;

    int* x = (int*)malloc(N * sizeof(int));
    int* y = (int*)malloc(N * sizeof(int));
    int host_expected_result = 0;
    for (int i = 0; i < N; i++) {
        x[i] = i % 10;
        y[i] = i % 3 - 1;
        host_expected_result += x[i] * y[i];
    }

    // step 05
    int result = 0;
    int *dx, *dy, *dz;
    int *z;







    // checking results
    if(host_expected_result == result) {
        std::cout << "Success" << std::endl;
    } else {
        std::cout << "Error" << std::endl;
        std::cout << "  expected: " << host_expected_result << std::endl;
        std::cout << "  got: " << result << std::endl;
    }

    free(x);
    free(y);

  return 0;
}